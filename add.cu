
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void add(int n, float x[], float y[])
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i+= stride) {
    y[i] = x[i] + y[i];
  }
}

int main()
{
  const int N = 1e6;
  float *x, *y;
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  add<<<1, 1024>>>(N, x, y);

  hipDeviceSynchronize();

  float maxErr = 0.0f;
  for (int i = 0; i < N; i++) {
    maxErr = max(maxErr, abs(y[i] - 3.0f));
  }
  cout << maxErr << endl;

  hipFree(x);
  hipFree(y);

  return 0;
}
