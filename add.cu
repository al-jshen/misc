
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void init(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
}

__global__ void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i+= stride) {
    y[i] = x[i] + y[i];
  }
}

int main()
{
  const int N = 1 << 20;
  float *x, *y;
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  /* for (int i = 0; i < N; i++) { */
  /*   x[i] = 1.0f; */
  /*   y[i] = 2.0f; */
  /* } */

  int blockSize = 512;
  int numBlocks = (N + blockSize - 1) / blockSize;
  /* cout << "blockSize" << blockSize << endl; */
  /* cout << "numBlocks" << numBlocks << endl; */
  init<<<numBlocks, blockSize>>>(N, x, y);
  add<<<numBlocks, blockSize>>>(N, x, y);

  hipDeviceSynchronize();

  float maxErr = 0.0f;
  for (int i = 0; i < N; i++) {
    maxErr = max(maxErr, abs(y[i] - 3.0f));
  }
  cout << maxErr << endl;

  hipFree(x);
  hipFree(y);

  return 0;
}
